#include "hip/hip_runtime.h"
#include "CudaLIC.h"

#include <iostream>
#include <stdio.h>
#include <float.h>
#include <hip/hip_fp16.h> // for CUDA 9.2

surface<void, cudaSurfaceType2D> velSurface;
__constant__ float3 planePoint;
__constant__ float3 planeNormal;
__constant__ float3 planeRight;
__constant__ float3 planeUp;
__constant__ float3 planeRightNorm;
__constant__ float3 planeUpNorm;
__constant__ float invBasisMat[9];
__constant__ float basisLength;

surface<void, cudaSurfaceType2D> outSurface;

texture<float,2> noiseTex;

__constant__ float texXMin;
__constant__ float texXMax;
__constant__ float texYMin;
__constant__ float texYMax;

void setVelSurfaceRef(hipArray* array)
{
    cudaBindSurfaceToArray(velSurface, array);
}

void setOutSurfaceRef(hipArray* array)
{
    cudaBindSurfaceToArray(outSurface, array);
}

void setPlaneConsts(void * point, void * normal, void * right, void * up, void * rightNorm, void * upNorm, void * basisMat, void * bLength)
{
    hipMemcpyToSymbol(HIP_SYMBOL(planePoint),point,sizeof(float3));
    hipMemcpyToSymbol(HIP_SYMBOL(planeNormal),normal,sizeof(float3));
    hipMemcpyToSymbol(HIP_SYMBOL(planeRight),right,sizeof(float3));
    hipMemcpyToSymbol(HIP_SYMBOL(planeUp),up,sizeof(float3));
    hipMemcpyToSymbol(HIP_SYMBOL(planeRightNorm),rightNorm,sizeof(float3));
    hipMemcpyToSymbol(HIP_SYMBOL(planeUpNorm),upNorm,sizeof(float3));
    hipMemcpyToSymbol(HIP_SYMBOL(invBasisMat),basisMat,9*sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(basisLength),bLength,sizeof(float));

    /*float devptr[9];

    // verify
    float * hostptr = (float*)rightNorm;
    hipMemcpyFromSymbol(devptr,HIP_SYMBOL(planeRightNorm),sizeof(float3));
    std::cerr << "Host rightpointN x: " << hostptr[0] << " y: " << hostptr[1] << " z: " << hostptr[2] << std::endl;
    std::cerr << "Dev rightpointN x: " << devptr[0] << " y: " << devptr[1] << " z: " << devptr[2] << std::endl;*/

}

void setTexConsts(void * xMin, void * xMax, void * yMin, void * yMax)
{
    hipMemcpyToSymbol(HIP_SYMBOL(texXMin),xMin,sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(texXMax),xMax,sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(texYMin),yMin,sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(texYMax),yMax,sizeof(float));

    //std::cerr << "Setting basis range X: " << ((float*)xMin)[0] << " " << ((float*)xMax)[0] << " Y: " << ((float*)yMin)[0] << " " << ((float*)yMax)[0] << std::endl;
}

void launchVel(uint4 * indices, float3 * verts, float3 * velocity, unsigned int * tetList, int numTets, int width, int height)
{
    //std::cerr << "Running velocity kernel" << std::endl;
    /*float devptr[9];
    hipMemcpyFromSymbol(devptr,HIP_SYMBOL(planePoint),sizeof(float3));
    std::cerr << "Dev planepoint x: " << devptr[0] << " y: " << devptr[1] << " z: " << devptr[2] << std::endl;*/

    dim3 blockc(8,8,1);
    dim3 gridc(width/8,height/8,1);

    velClear<<< gridc, blockc >>>(width,height);
    hipDeviceSynchronize();

    int threadsPerBlock = 256;
    dim3 block(threadsPerBlock,1,1);

    int critcalPoints = width * height * 1.1;
    int pointsPerTet = critcalPoints * 1.2 / numTets;
    float blocksPerTet = pointsPerTet / (threadsPerBlock * 20.0f);
    
    int griddim;
    //griddim = (numTets / 256) + 1;

    bool threadPerTet = false;

    int sharedMemPerBlock = 0;
    int tetsPerBlock = 1;

    if(blocksPerTet >= 1.0f)
    {
	griddim = ((int)blocksPerTet) * numTets;
	sharedMemPerBlock = 24 * sizeof(float);
    }
    else
    {
	float tetsPerBlockf = 1.0f / blocksPerTet;

	// may need to lower limit if reg count goes down
	if(tetsPerBlockf > 64.0f)
	{
	    griddim = (numTets / 256) + 1;
	    threadPerTet = true;
	}
	else if(tetsPerBlockf > 32.0f)
	{
	    griddim = (numTets / 64) + 1;
	    tetsPerBlock = 64;
	}
	else if(tetsPerBlockf > 16.0f)
	{
	    griddim = (numTets / 32) + 1;
	    tetsPerBlock = 32;
	}
	else if(tetsPerBlockf > 8.0f)
	{
	    griddim = (numTets / 16) + 1;
	    tetsPerBlock = 16;
	}
	else if(tetsPerBlockf > 4.0f)
	{
	    griddim = (numTets / 8) + 1;
	    tetsPerBlock = 8;
	}
	else if(tetsPerBlockf > 2.0f)
	{
	    griddim = (numTets / 4) + 1;
	    tetsPerBlock = 4;
	}
	else
	{
	    griddim = (numTets / 2  ) + 1;
	    tetsPerBlock = 2;
	}
	sharedMemPerBlock = 24 * sizeof(float) * tetsPerBlock;
    }

    //std::cerr << "NumTets: " << numTets << " Grid dim: " << griddim << std::endl;
    dim3 grid(griddim,1,1);

    if(threadPerTet)
    {
	//std::cerr << "Using single thread kernel" << std::endl;
	velKernel<<< grid, block >>>(indices,verts,velocity,tetList,numTets,width,height,width/2.0,height/2.0);
    }
    else
    {
	//std::cerr << "Using multi thread kernel, blocksPerTet: " << (int)blocksPerTet << " tetsPerBlock: " << tetsPerBlock << " smem: " << sharedMemPerBlock << std::endl;
	velSplitKernel<<< grid, block, sharedMemPerBlock >>>(indices,verts,velocity,tetList,numTets,(int)blocksPerTet,tetsPerBlock,width,height,width/2.0,height/2.0);
    }
}

void launchLIC(int width, int height, float length, hipArray * noiseArray)
{
    //std::cerr << "Running LIC kernel" << std::endl;
    hipBindTextureToArray(noiseTex,noiseArray);

    dim3 blockc(8,8,1);
    dim3 gridc(width/8,height/8,1);
    //dim3 gridc(1,1,1);

    licKernel<<< gridc, blockc >>>(width,height,length);
    //licKernel<<< gridc, blockc >>>(1024,1024,length);
    hipDeviceSynchronize();

    hipUnbindTexture(noiseTex);
}

void launchMakeTetList(unsigned int * tetList, unsigned int * numTets, int totalTets, uint4 * indices, float3 * verts)
{
    dim3 block(256,1,1);
    int griddim = (totalTets / 256) + 1;
    //std::cerr << "NumTets: " << numTets << " Grid dim: " << griddim << std::endl;
    dim3 grid(griddim,1,1);

    makeTetListKernel<<< grid, block >>>(tetList,numTets,totalTets,indices,verts);
}

__global__ void velClear(int width, int height) 
{
    // Calculate surface coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        unsigned short data[2];
	data[0] = __float2half_rn(0.0);
	data[1] = __float2half_rn(0.0);
        // Read from input surface
        //surf2Dread(&data,  inputSurfRef, x * 4, y);
        // Write to output surface
        surf2Dwrite(*((uchar4*)data), velSurface, x * 4, y);
    }
}

#define VEL_EPS 0.001f

__global__ void velKernel(uint4 * ind, float3 * verts, float3 * velocity, unsigned int * tetList, int numTets, int width, int height, float hwidth, float hheight)
{
    int tetid = blockIdx.x*blockDim.x + threadIdx.x;
    if(tetid >= numTets)
    {
	return;
    }

    tetid = tetList[tetid];

    /*if(tetid == 0)
    {
	printf("PlaneUp %f %f %f\n",planeUp.x,planeUp.y,planeUp.z);
	printf("PlaneUpN %f %f %f\n",planeUpNorm.x,planeUpNorm.y,planeUpNorm.z);
	printf("PlaneRight %f %f %f\n",planeRight.x,planeRight.y,planeRight.z);
	printf("PlaneRightN %f %f %f\n",planeRightNorm.x,planeRightNorm.y,planeRightNorm.z);
	printf("PlanePoint %f %f %f\n",planePoint.x,planePoint.y,planePoint.z);
	printf("PlaneNormal %f %f %f\n",planeNormal.x,planeNormal.y,planeNormal.z);
	printf("BasisMat: %f %f %f %f %f %f %f %f %f\n",invBasisMat[0],invBasisMat[1],invBasisMat[2],invBasisMat[3],invBasisMat[4],invBasisMat[5],invBasisMat[6],invBasisMat[7],invBasisMat[8]);
    }*/

    float4 tetpoints[4];
    *((float3*)&tetpoints[0]) = verts[ind[tetid].x];
    *((float3*)&tetpoints[1]) = verts[ind[tetid].y];
    *((float3*)&tetpoints[2]) = verts[ind[tetid].z];
    *((float3*)&tetpoints[3]) = verts[ind[tetid].w];

    //printf("Point0 %f %f %f\n",tetpoints[0].x,tetpoints[0].y,tetpoints[0].z);
    //printf("Point1 %f %f %f\n",tetpoints[1].x,tetpoints[1].y,tetpoints[1].z);
    //printf("Point2 %f %f %f\n",tetpoints[2].x,tetpoints[2].y,tetpoints[2].z);
    //printf("Point3 %f %f %f\n",tetpoints[3].x,tetpoints[3].y,tetpoints[3].z);

    // find viewing plane distance
    tetpoints[0].w = (tetpoints[0].x - planePoint.x) * planeNormal.x + (tetpoints[0].y - planePoint.y) * planeNormal.y + (tetpoints[0].z - planePoint.z) * planeNormal.z;
    tetpoints[1].w = (tetpoints[1].x - planePoint.x) * planeNormal.x + (tetpoints[1].y - planePoint.y) * planeNormal.y + (tetpoints[1].z - planePoint.z) * planeNormal.z;
    tetpoints[2].w = (tetpoints[2].x - planePoint.x) * planeNormal.x + (tetpoints[2].y - planePoint.y) * planeNormal.y + (tetpoints[2].z - planePoint.z) * planeNormal.z;
    tetpoints[3].w = (tetpoints[3].x - planePoint.x) * planeNormal.x + (tetpoints[3].y - planePoint.y) * planeNormal.y + (tetpoints[3].z - planePoint.z) * planeNormal.z;

    //printf("Distances: %f %f %f %f\n",tetpoints[0].w,tetpoints[1].w,tetpoints[2].w,tetpoints[3].w);

    // determine if plane passes through tet
    /*int count = 0;
    if(tetpoints[0].w > 0.0f)
    {
	count++;
    }
    if(tetpoints[1].w > 0.0f)
    {
	count++;
    }
    if(tetpoints[2].w > 0.0f)
    {
	count++;
    }
    if(tetpoints[3].w > 0.0f)
    {
	count++;
    }

    if(count == 0 || count == 4)
    {
	return;
    }*/

    // project points onto plane and find basis values
    float3 projpoint;
    float2 basisMin;
    float2 basisMax;
    float tempx;
    projpoint.x = tetpoints[0].x - (planeNormal.x * tetpoints[0].w) - planePoint.x;
    projpoint.y = tetpoints[0].y - (planeNormal.y * tetpoints[0].w) - planePoint.y;
    projpoint.z = tetpoints[0].z - (planeNormal.z * tetpoints[0].w) - planePoint.z;

    //printf("projpoint: %f %f %f\n",projpoint.x,projpoint.y,projpoint.z);

    //tempx = projpoint.x * invBasisMat[0] + projpoint.y * invBasisMat[3] + projpoint.z * invBasisMat[6];
    //projpoint.y = projpoint.x * invBasisMat[1] + projpoint.y * invBasisMat[4] + projpoint.z * invBasisMat[7];
    tempx = (projpoint.x * planeUpNorm.x + projpoint.y * planeUpNorm.y + projpoint.z * planeUpNorm.z) / basisLength;
    projpoint.y = (projpoint.x * planeRightNorm.x + projpoint.y * planeRightNorm.y + projpoint.z * planeRightNorm.z) / basisLength;

    //printf("basis %f %f\n",tempx,projpoint.y);

    basisMin.x = basisMax.x = tempx;
    basisMin.y = basisMax.y = projpoint.y;
    
    projpoint.x = tetpoints[1].x - (planeNormal.x * tetpoints[1].w) - planePoint.x;
    projpoint.y = tetpoints[1].y - (planeNormal.y * tetpoints[1].w) - planePoint.y;
    projpoint.z = tetpoints[1].z - (planeNormal.z * tetpoints[1].w) - planePoint.z;

    //tempx = projpoint.x * invBasisMat[0] + projpoint.y * invBasisMat[3] + projpoint.z * invBasisMat[6];
    //projpoint.y = projpoint.x * invBasisMat[1] + projpoint.y * invBasisMat[4] + projpoint.z * invBasisMat[7];
    tempx = (projpoint.x * planeUpNorm.x + projpoint.y * planeUpNorm.y + projpoint.z * planeUpNorm.z) / basisLength;
    projpoint.y = (projpoint.x * planeRightNorm.x + projpoint.y * planeRightNorm.y + projpoint.z * planeRightNorm.z) / basisLength;


    //printf("basis %f %f\n",tempx,projpoint.y);

    basisMin.x = fminf(basisMin.x,tempx);
    basisMax.x = fmaxf(basisMax.x,tempx);
    basisMin.y = fminf(basisMin.y,projpoint.y);
    basisMax.y = fmaxf(basisMax.y,projpoint.y);

    projpoint.x = tetpoints[2].x - (planeNormal.x * tetpoints[2].w) - planePoint.x;
    projpoint.y = tetpoints[2].y - (planeNormal.y * tetpoints[2].w) - planePoint.y;
    projpoint.z = tetpoints[2].z - (planeNormal.z * tetpoints[2].w) - planePoint.z;

    //tempx = projpoint.x * invBasisMat[0] + projpoint.y * invBasisMat[3] + projpoint.z * invBasisMat[6];
    //projpoint.y = projpoint.x * invBasisMat[1] + projpoint.y * invBasisMat[4] + projpoint.z * invBasisMat[7];
    tempx = (projpoint.x * planeUpNorm.x + projpoint.y * planeUpNorm.y + projpoint.z * planeUpNorm.z) / basisLength;
    projpoint.y = (projpoint.x * planeRightNorm.x + projpoint.y * planeRightNorm.y + projpoint.z * planeRightNorm.z) / basisLength;

    //printf("basis %f %f\n",tempx,projpoint.y);

    basisMin.x = fminf(basisMin.x,tempx);
    basisMax.x = fmaxf(basisMax.x,tempx);
    basisMin.y = fminf(basisMin.y,projpoint.y);
    basisMax.y = fmaxf(basisMax.y,projpoint.y);

    projpoint.x = tetpoints[3].x - (planeNormal.x * tetpoints[3].w) - planePoint.x;
    projpoint.y = tetpoints[3].y - (planeNormal.y * tetpoints[3].w) - planePoint.y;
    projpoint.z = tetpoints[3].z - (planeNormal.z * tetpoints[3].w) - planePoint.z;

    //tempx = projpoint.x * invBasisMat[0] + projpoint.y * invBasisMat[3] + projpoint.z * invBasisMat[6];
    //projpoint.y = projpoint.x * invBasisMat[1] + projpoint.y * invBasisMat[4] + projpoint.z * invBasisMat[7];
    tempx = (projpoint.x * planeUpNorm.x + projpoint.y * planeUpNorm.y + projpoint.z * planeUpNorm.z) / basisLength;
    projpoint.y = (projpoint.x * planeRightNorm.x + projpoint.y * planeRightNorm.y + projpoint.z * planeRightNorm.z) / basisLength;

    //printf("basis %f %f\n",tempx,projpoint.y);

    basisMin.x = fminf(basisMin.x,tempx);
    basisMax.x = fmaxf(basisMax.x,tempx);
    basisMin.y = fminf(basisMin.y,projpoint.y);
    basisMax.y = fmaxf(basisMax.y,projpoint.y);

    //printf("X: %f %f\n",basisMin.x,basisMax.x);
    //printf("Y: %f %f\n",basisMin.y,basisMax.y);

    /*basisMin.x = ceilf(basisMin.x - 0.5f) + 0.5f;
    basisMin.y = ceilf(basisMin.y - 0.5f) + 0.5f;
    basisMax.x = floorf(basisMax.x - 0.5f) + 0.500001f;
    basisMax.y = floorf(basisMax.y - 0.5f) + 0.500001f;*/

    basisMin.x = floorf(basisMin.x - 0.5f) - 0.5f;
    basisMin.y = floorf(basisMin.y - 0.5f) - 0.5f;
    basisMax.x = ceilf(basisMax.x + 0.5f) + 0.500001f;
    basisMax.y = ceilf(basisMax.y + 0.5f) + 0.500001f;

    //printf("X: %f %f\n",basisMin.x,basisMax.x);
    //printf("Y: %f %f\n",basisMin.y,basisMax.y);

    if(basisMin.x > basisMax.x || basisMin.y > basisMax.y)
    {
	//printf("Basis range exit x: %f %f y: %f %f\n",basisMin.x,basisMax.x,basisMin.y,basisMax.y);
	// no critical points in tet
	return;
    }

    //find if tet is outside the basis bounds
    if(basisMin.x > texXMax || basisMax.x < texXMin || basisMin.y > texYMax || basisMax.y < texYMin)
    {
	//printf("X: %f %f\n",basisMin.x,basisMax.x);
	//printf("Y: %f %f\n",basisMin.y,basisMax.y);
	return;
    }

    /*if(tetid < 1024*1024)
    {
	unsigned short data[2];
	data[0] = __float2half_rn(0.5);
	data[1] = __float2half_rn(0.5);
	surf2Dwrite(*((uchar4*)data), velSurface, (tetid % 1024) * 4, tetid / 1024);
    }*/

    /*if(basisMin.x >= -512 && basisMax.x <= 512 && basisMin.y >= -512 && basisMax.y <= 512)
    {
	if(tetid < 1024*1024)
		{
		    unsigned short data[2];
		    data[0] = __float2half_rn(1.0);
		    data[1] = __float2half_rn(0.0);
		    surf2Dwrite(*((uchar4*)data), velSurface, (tetid % 1024) * 4, tetid / 1024);
		}
    }
    else
    {
	if(tetid < 1024*1024)
		{
		    unsigned short data[2];
		    data[0] = __float2half_rn(0.5);
		    data[1] = __float2half_rn(0.5);
		    surf2Dwrite(*((uchar4*)data), velSurface, (tetid % 1024) * 4, tetid / 1024);
		}
    }*/

    //return;

    // create matrix to solve for barycentric coords

    // create matrix rows to inverse
    tetpoints[0].x = tetpoints[0].x - tetpoints[3].x;
    tetpoints[0].y = tetpoints[0].y - tetpoints[3].y;
    tetpoints[0].z = tetpoints[0].z - tetpoints[3].z;
    tetpoints[1].x = tetpoints[1].x - tetpoints[3].x;
    tetpoints[1].y = tetpoints[1].y - tetpoints[3].y;
    tetpoints[1].z = tetpoints[1].z - tetpoints[3].z;
    tetpoints[2].x = tetpoints[2].x - tetpoints[3].x;
    tetpoints[2].y = tetpoints[2].y - tetpoints[3].y;
    tetpoints[2].z = tetpoints[2].z - tetpoints[3].z;
    
    // matrix determinant
    float det = tetpoints[0].x*tetpoints[1].y*tetpoints[2].z + tetpoints[1].x*tetpoints[2].y*tetpoints[0].z + tetpoints[2].x*tetpoints[0].y*tetpoints[1].z - tetpoints[2].x*tetpoints[1].y*tetpoints[0].z - tetpoints[1].x*tetpoints[0].y*tetpoints[2].z - tetpoints[0].x*tetpoints[2].y*tetpoints[1].z;

    // invert
    float tetMat[9];
    tetMat[0] = (tetpoints[2].z*tetpoints[1].y - tetpoints[1].z*tetpoints[2].y) / det;
    tetMat[1] = (tetpoints[2].z*tetpoints[1].x - tetpoints[1].z*tetpoints[2].x) / -det;
    tetMat[2] = (tetpoints[2].y*tetpoints[1].x - tetpoints[1].y*tetpoints[2].x) / det;
    tetMat[3] = (tetpoints[2].z*tetpoints[0].y - tetpoints[0].z*tetpoints[2].y) / -det;
    tetMat[4] = (tetpoints[2].z*tetpoints[0].x - tetpoints[0].z*tetpoints[2].x) / det;
    tetMat[5] = (tetpoints[2].y*tetpoints[0].x - tetpoints[0].y*tetpoints[2].x) / -det;
    tetMat[6] = (tetpoints[1].z*tetpoints[0].y - tetpoints[0].z*tetpoints[1].y) / det;
    tetMat[7] = (tetpoints[1].z*tetpoints[0].x - tetpoints[0].z*tetpoints[1].x) / -det;
    tetMat[8] = (tetpoints[1].y*tetpoints[0].x - tetpoints[0].y*tetpoints[1].x) / det;

    //printf("barMat:\n%f %f %f\n%f %f %f\n%f %f %f\n",tetMat[0],tetMat[1],tetMat[2],tetMat[3],tetMat[4],tetMat[5],tetMat[6],tetMat[7],tetMat[8]);

    // get tet point velocities
     float3 tetVel[4];
    tetVel[0] = velocity[ind[tetid].x];
    tetVel[1] = velocity[ind[tetid].y];
    tetVel[2] = velocity[ind[tetid].z];
    tetVel[3] = velocity[ind[tetid].w];

    //float tests = (basisMax.x - basisMin.x) * (basisMax.y - basisMin.y);
    //printf("Critical Points: %f\n",tests);

    // process all critical points
    for(float i = basisMin.x; i <= basisMax.x; i = i + 1.0f)
    {
	for(float j = basisMin.y; j <= basisMax.y; j = j + 1.0f)
	{
	    // find barycentric coords
	    float3 tempPoint;

	    tempPoint.x = i * planeUp.x + j * planeRight.x + planePoint.x - tetpoints[3].x;
	    tempPoint.y = i * planeUp.y + j * planeRight.y + planePoint.y - tetpoints[3].y;
	    tempPoint.z = i * planeUp.z + j * planeRight.z + planePoint.z - tetpoints[3].z;

	    //printf("temppoint %f %f %f\n",tempPoint.x,tempPoint.y,tempPoint.z);

	    float4 coords;
	    coords.x = tempPoint.x * tetMat[0] + tempPoint.y * tetMat[1] + tempPoint.z * tetMat[2];
	    coords.y = tempPoint.x * tetMat[3] + tempPoint.y * tetMat[4] + tempPoint.z * tetMat[5];
	    coords.z = tempPoint.x * tetMat[6] + tempPoint.y * tetMat[7] + tempPoint.z * tetMat[8];
	    coords.w = 1.0f - coords.x - coords.y - coords.z;

	    //printf("Coords %f %f %f %f\n",coords.x,coords.y,coords.z,coords.w);

	    if(coords.x > 1.0f + VEL_EPS || coords.x < 0.0f - VEL_EPS || coords.y > 1.0f + VEL_EPS || coords.y < 0.0f - VEL_EPS || coords.z > 1.0f + VEL_EPS || coords.z < 0.0f - VEL_EPS || coords.w > 1.0f + VEL_EPS || coords.w < 0.0f - VEL_EPS)
	    {	
		continue;
	    }

	    /*if(tetid < 1024*1024)
	    {
		unsigned short data[2];
		data[0] = __float2half_rn(0.0);
		data[1] = __float2half_rn(1.0);
		surf2Dwrite(*((uchar4*)data), velSurface, (tetid % 1024) * 4, tetid / 1024);
	    }*/

	    int2 texelIndex;
	    texelIndex.x = lrintf((floorf(j) + hwidth));
	    texelIndex.y = lrintf(floor(i) + hheight);
	    //printf("Texel index: %f %f %d %d\n",j,i,texelIndex.x,texelIndex.y);
	    if(texelIndex.x >= 0 && texelIndex.x < width && texelIndex.y >= 0 && texelIndex.y < height)
	    {
		/*if(tetid < 1024*1024)
		{
		    unsigned short data[2];
		    data[0] = __float2half_rn(0.0);
		    data[1] = __float2half_rn(1.0);
		    surf2Dwrite(*((uchar4*)data), velSurface, (tetid % 1024) * 4, tetid / 1024);
		}*/

		// find point velocity
		float3 myVelocity;
		myVelocity.x = coords.x * tetVel[0].x + coords.y * tetVel[1].x + coords.z * tetVel[2].x + coords.w * tetVel[3].x;
		myVelocity.y = coords.x * tetVel[0].y + coords.y * tetVel[1].y + coords.z * tetVel[2].y + coords.w * tetVel[3].y;
		myVelocity.z = coords.x * tetVel[0].z + coords.y * tetVel[1].z + coords.z * tetVel[2].z + coords.w * tetVel[3].z;

		// also temp x
		float mag;
		mag = planeRightNorm.x * myVelocity.x + planeRightNorm.y * myVelocity.y + planeRightNorm.z * myVelocity.z;
		myVelocity.y = planeUpNorm.x * myVelocity.x + planeUpNorm.y * myVelocity.y + planeUpNorm.z * myVelocity.z;
		myVelocity.x = mag;

		// normalize
		mag = myVelocity.x * myVelocity.x + myVelocity.y * myVelocity.y;
		mag = sqrt(mag);
		if(mag > 0.0f)
		{
		    myVelocity.x = myVelocity.x / mag;
		    myVelocity.y = myVelocity.y / mag;
		}

		unsigned short data[2];
		// project onto texture basis
		/*float2 output;
		output.x = planeRightNorm.x * myVelocity.x + planeRightNorm.y * myVelocity.y + planeRightNorm.z * myVelocity.z;
		output.y = planeUpNorm.x * myVelocity.x + planeUpNorm.y * myVelocity.y + planeUpNorm.z * myVelocity.z;
		printf("Output x: %d y: %d %f %f\n",texelIndex.x,texelIndex.y,output.x,output.y);
		data[0] = __float2half_rn(output.x);
		data[1] = __float2half_rn(output.y);*/
		data[0] = __float2half_rn(myVelocity.x);
		data[1] = __float2half_rn(myVelocity.y);
		/*if(output.x != 0.0f || output.y != 0.0f)
		{
		    data[0] = __float2half_rn(1.0f);
		    data[1] = __float2half_rn(0.0f);
		}
		else
		{
		    data[0] = __float2half_rn(0.0f);
		    data[1] = __float2half_rn(1.0f);
		    printf("Coords %f %f %f %f\n",coords.x,coords.y,coords.z,coords.w);
		    printf("Vel %f %f %f\n",tetVel[0].x,tetVel[0].y,tetVel[0].z);
		    printf("Vel %f %f %f\n",tetVel[1].x,tetVel[1].y,tetVel[1].z);
		    printf("Vel %f %f %f\n",tetVel[2].x,tetVel[2].y,tetVel[2].z);
		    printf("Vel %f %f %f\n",tetVel[3].x,tetVel[3].y,tetVel[3].z);
		}*/
		surf2Dwrite(*((uchar4*)data), velSurface, texelIndex.x * 4, texelIndex.y);
	    }
	}
    }
}

__global__ void velSplitKernel(uint4 * ind, float3 * verts, float3 * velocity, unsigned int * tetList, int numTets, int blocksPerTet, int tetsPerBlock, int width, int height, float hwidth, float hheight)
{
    extern __shared__ float3 tetData[];

    int myOffset;
    int blockOffset;
    int threadOffset;
    int threadsPerTet;

    if(blocksPerTet == 0)
    {
	blockOffset = blockIdx.x * tetsPerBlock;
	myOffset = threadIdx.x / (blockDim.x / tetsPerBlock);
	threadOffset = threadIdx.x % (blockDim.x / tetsPerBlock);
	threadsPerTet = (blockDim.x / tetsPerBlock);
    }
    else
    {
	blockOffset = blockIdx.x / blocksPerTet;
	myOffset = 0;
	threadOffset = (blockIdx.x % blocksPerTet) * blockDim.x + threadIdx.x;
	threadsPerTet = blockDim.x * blocksPerTet;
    }

    //printf("myOffset %d, blockOffset %d, threadOffset %d, threadsPerTet %d, blocksPerTet %d, tetsPerBlock %d, thread %d, block %d\n",myOffset,blockOffset,threadOffset,threadsPerTet,blocksPerTet,tetsPerBlock,threadIdx.x,blockIdx.x);

    if(blockOffset + myOffset >= numTets)
    {
	return;
    }

    int tetid = tetList[blockOffset + myOffset];

    myOffset = myOffset * 4;

    // read tet data
    if(threadOffset == 0 || threadIdx.x == 0)
    {
	int toffset = myOffset;
	tetData[toffset] = verts[ind[tetid].x];
	tetData[toffset+1] = verts[ind[tetid].y];
	tetData[toffset+2] = verts[ind[tetid].z];
	tetData[toffset+3] = verts[ind[tetid].w];

	toffset = toffset + 4 * tetsPerBlock;
	tetData[toffset] = velocity[ind[tetid].x];
	tetData[toffset+1] = velocity[ind[tetid].y];
	tetData[toffset+2] = velocity[ind[tetid].z];
	tetData[toffset+3] = velocity[ind[tetid].w];
    }

    __syncthreads();

    //int velind = myOffset + 4 * tetsPerBlock;
    //printf("Vert %f %f %f, vel %f %f %f\n",tetData[myOffset].x,tetData[myOffset].y,tetData[myOffset].z,tetData[velind].x,tetData[velind].y,tetData[velind].z);

    float dist[4];

    // find viewing plane distance
    dist[0] = (tetData[myOffset+0].x - planePoint.x) * planeNormal.x + (tetData[myOffset+0].y - planePoint.y) * planeNormal.y + (tetData[myOffset+0].z - planePoint.z) * planeNormal.z;
    dist[1] = (tetData[myOffset+1].x - planePoint.x) * planeNormal.x + (tetData[myOffset+1].y - planePoint.y) * planeNormal.y + (tetData[myOffset+1].z - planePoint.z) * planeNormal.z;
    dist[2] = (tetData[myOffset+2].x - planePoint.x) * planeNormal.x + (tetData[myOffset+2].y - planePoint.y) * planeNormal.y + (tetData[myOffset+2].z - planePoint.z) * planeNormal.z;
    dist[3] = (tetData[myOffset+3].x - planePoint.x) * planeNormal.x + (tetData[myOffset+3].y - planePoint.y) * planeNormal.y + (tetData[myOffset+3].z - planePoint.z) * planeNormal.z;

    //printf("dist %f %f %f %f\n",dist[0],dist[1],dist[2],dist[3]);

    // project points onto plane and find basis values
    float3 projpoint;
    float2 basisMin;
    float2 basisMax;
    float tempx;
    projpoint.x = tetData[myOffset+0].x - (planeNormal.x * dist[0]) - planePoint.x;
    projpoint.y = tetData[myOffset+0].y - (planeNormal.y * dist[0]) - planePoint.y;
    projpoint.z = tetData[myOffset+0].z - (planeNormal.z * dist[0]) - planePoint.z;

    tempx = (projpoint.x * planeUpNorm.x + projpoint.y * planeUpNorm.y + projpoint.z * planeUpNorm.z) / basisLength;
    projpoint.y = (projpoint.x * planeRightNorm.x + projpoint.y * planeRightNorm.y + projpoint.z * planeRightNorm.z) / basisLength;

    basisMin.x = basisMax.x = tempx;
    basisMin.y = basisMax.y = projpoint.y;
    
    projpoint.x = tetData[myOffset+1].x - (planeNormal.x * dist[1]) - planePoint.x;
    projpoint.y = tetData[myOffset+1].y - (planeNormal.y * dist[1]) - planePoint.y;
    projpoint.z = tetData[myOffset+1].z - (planeNormal.z * dist[1]) - planePoint.z;

    tempx = (projpoint.x * planeUpNorm.x + projpoint.y * planeUpNorm.y + projpoint.z * planeUpNorm.z) / basisLength;
    projpoint.y = (projpoint.x * planeRightNorm.x + projpoint.y * planeRightNorm.y + projpoint.z * planeRightNorm.z) / basisLength;

    basisMin.x = fminf(basisMin.x,tempx);
    basisMax.x = fmaxf(basisMax.x,tempx);
    basisMin.y = fminf(basisMin.y,projpoint.y);
    basisMax.y = fmaxf(basisMax.y,projpoint.y);

    projpoint.x = tetData[myOffset+2].x - (planeNormal.x * dist[2]) - planePoint.x;
    projpoint.y = tetData[myOffset+2].y - (planeNormal.y * dist[2]) - planePoint.y;
    projpoint.z = tetData[myOffset+2].z - (planeNormal.z * dist[2]) - planePoint.z;

    tempx = (projpoint.x * planeUpNorm.x + projpoint.y * planeUpNorm.y + projpoint.z * planeUpNorm.z) / basisLength;
    projpoint.y = (projpoint.x * planeRightNorm.x + projpoint.y * planeRightNorm.y + projpoint.z * planeRightNorm.z) / basisLength;

    basisMin.x = fminf(basisMin.x,tempx);
    basisMax.x = fmaxf(basisMax.x,tempx);
    basisMin.y = fminf(basisMin.y,projpoint.y);
    basisMax.y = fmaxf(basisMax.y,projpoint.y);

    projpoint.x = tetData[myOffset+3].x - (planeNormal.x * dist[3]) - planePoint.x;
    projpoint.y = tetData[myOffset+3].y - (planeNormal.y * dist[3]) - planePoint.y;
    projpoint.z = tetData[myOffset+3].z - (planeNormal.z * dist[3]) - planePoint.z;

    tempx = (projpoint.x * planeUpNorm.x + projpoint.y * planeUpNorm.y + projpoint.z * planeUpNorm.z) / basisLength;
    projpoint.y = (projpoint.x * planeRightNorm.x + projpoint.y * planeRightNorm.y + projpoint.z * planeRightNorm.z) / basisLength;

    basisMin.x = fminf(basisMin.x,tempx);
    basisMax.x = fmaxf(basisMax.x,tempx);
    basisMin.y = fminf(basisMin.y,projpoint.y);
    basisMax.y = fmaxf(basisMax.y,projpoint.y);

    //printf("basisX %f %f, basisY %f %f, bxtex %f %f, bytex %f %f\n",basisMin.x,basisMax.x,basisMin.y,basisMax.y,texXMin,texXMax,texYMin,texYMax);

    // area larger than needed for rounding errors
    basisMin.x = floorf(basisMin.x - 0.5f) - 0.5f;
    basisMin.y = floorf(basisMin.y - 0.5f) - 0.5f;
    basisMax.x = ceilf(basisMax.x + 0.5f) + 0.500001f;
    basisMax.y = ceilf(basisMax.y + 0.5f) + 0.500001f;

    if(basisMin.x > basisMax.x || basisMin.y > basisMax.y)
    {
	// no critical points in tet
	return;
    }

    //find if tet is outside the basis bounds
    if(basisMin.x > texXMax || basisMax.x < texXMin || basisMin.y > texYMax || basisMax.y < texYMin)
    {
	return;
    }

    // create matrix to solve for barycentric coords

    // create matrix rows to inverse
    tetData[myOffset+0].x = tetData[myOffset+0].x - tetData[myOffset+3].x;
    tetData[myOffset+0].y = tetData[myOffset+0].y - tetData[myOffset+3].y;
    tetData[myOffset+0].z = tetData[myOffset+0].z - tetData[myOffset+3].z;
    tetData[myOffset+1].x = tetData[myOffset+1].x - tetData[myOffset+3].x;
    tetData[myOffset+1].y = tetData[myOffset+1].y - tetData[myOffset+3].y;
    tetData[myOffset+1].z = tetData[myOffset+1].z - tetData[myOffset+3].z;
    tetData[myOffset+2].x = tetData[myOffset+2].x - tetData[myOffset+3].x;
    tetData[myOffset+2].y = tetData[myOffset+2].y - tetData[myOffset+3].y;
    tetData[myOffset+2].z = tetData[myOffset+2].z - tetData[myOffset+3].z;
    
    // matrix determinant
    float det = tetData[myOffset+0].x*tetData[myOffset+1].y*tetData[myOffset+2].z + tetData[myOffset+1].x*tetData[myOffset+2].y*tetData[myOffset+0].z + tetData[myOffset+2].x*tetData[myOffset+0].y*tetData[myOffset+1].z - tetData[myOffset+2].x*tetData[myOffset+1].y*tetData[myOffset+0].z - tetData[myOffset+1].x*tetData[myOffset+0].y*tetData[myOffset+2].z - tetData[myOffset+0].x*tetData[myOffset+2].y*tetData[myOffset+1].z;

    // invert
    float tetMat[9];
    tetMat[0] = (tetData[myOffset+2].z*tetData[myOffset+1].y - tetData[myOffset+1].z*tetData[myOffset+2].y) / det;
    tetMat[1] = (tetData[myOffset+2].z*tetData[myOffset+1].x - tetData[myOffset+1].z*tetData[myOffset+2].x) / -det;
    tetMat[2] = (tetData[myOffset+2].y*tetData[myOffset+1].x - tetData[myOffset+1].y*tetData[myOffset+2].x) / det;
    tetMat[3] = (tetData[myOffset+2].z*tetData[myOffset+0].y - tetData[myOffset+0].z*tetData[myOffset+2].y) / -det;
    tetMat[4] = (tetData[myOffset+2].z*tetData[myOffset+0].x - tetData[myOffset+0].z*tetData[myOffset+2].x) / det;
    tetMat[5] = (tetData[myOffset+2].y*tetData[myOffset+0].x - tetData[myOffset+0].y*tetData[myOffset+2].x) / -det;
    tetMat[6] = (tetData[myOffset+1].z*tetData[myOffset+0].y - tetData[myOffset+0].z*tetData[myOffset+1].y) / det;
    tetMat[7] = (tetData[myOffset+1].z*tetData[myOffset+0].x - tetData[myOffset+0].z*tetData[myOffset+1].x) / -det;
    tetMat[8] = (tetData[myOffset+1].y*tetData[myOffset+0].x - tetData[myOffset+0].y*tetData[myOffset+1].x) / det;


    int2 basisRange;
    basisRange.x = (int)(basisMax.x - basisMin.x);
    basisRange.y = (int)(basisMax.y - basisMin.y);
    
    int velOffset = myOffset + 4 * tetsPerBlock;
    //printf("Brange %d %d, voffset %d\n",basisRange.x,basisRange.y,velOffset);
    while(1)
    {
	float2 basis;
	basis.x = basisMin.x + (float)(threadOffset % basisRange.x);
	basis.y = basisMin.y + (float)(threadOffset / basisRange.x);

	threadOffset = threadOffset + threadsPerTet;

	if(basis.y > basisMax.y)
	{
	    break;
	}

	// find barycentric coords
	float3 tempPoint;

	tempPoint.x = basis.x * planeUp.x + basis.y * planeRight.x + planePoint.x - tetData[myOffset+3].x;
	tempPoint.y = basis.x * planeUp.y + basis.y * planeRight.y + planePoint.y - tetData[myOffset+3].y;
	tempPoint.z = basis.x * planeUp.z + basis.y * planeRight.z + planePoint.z - tetData[myOffset+3].z;

	float4 coords;
	coords.x = tempPoint.x * tetMat[0] + tempPoint.y * tetMat[1] + tempPoint.z * tetMat[2];
	coords.y = tempPoint.x * tetMat[3] + tempPoint.y * tetMat[4] + tempPoint.z * tetMat[5];
	coords.z = tempPoint.x * tetMat[6] + tempPoint.y * tetMat[7] + tempPoint.z * tetMat[8];
	coords.w = 1.0f - coords.x - coords.y - coords.z;

	if(coords.x > 1.0f + VEL_EPS || coords.x < 0.0f - VEL_EPS || coords.y > 1.0f + VEL_EPS || coords.y < 0.0f - VEL_EPS || coords.z > 1.0f + VEL_EPS || coords.z < 0.0f - VEL_EPS || coords.w > 1.0f + VEL_EPS || coords.w < 0.0f - VEL_EPS)
	{	
	    continue;
	}

	int2 texelIndex;
	texelIndex.x = lrintf((floorf(basis.y) + hwidth));
	texelIndex.y = lrintf(floor(basis.x) + hheight);
	if(texelIndex.x >= 0 && texelIndex.x < width && texelIndex.y >= 0 && texelIndex.y < height)
	{
	    // find point velocity
	    float3 myVelocity;
	    myVelocity.x = coords.x * tetData[velOffset+0].x + coords.y * tetData[velOffset+1].x + coords.z * tetData[velOffset+2].x + coords.w * tetData[velOffset+3].x;
	    myVelocity.y = coords.x * tetData[velOffset+0].y + coords.y * tetData[velOffset+1].y + coords.z * tetData[velOffset+2].y + coords.w * tetData[velOffset+3].y;
	    myVelocity.z = coords.x * tetData[velOffset+0].z + coords.y * tetData[velOffset+1].z + coords.z * tetData[velOffset+2].z + coords.w * tetData[velOffset+3].z;

	    // also temp x
	    float mag;
	    mag = planeRightNorm.x * myVelocity.x + planeRightNorm.y * myVelocity.y + planeRightNorm.z * myVelocity.z;
	    myVelocity.y = planeUpNorm.x * myVelocity.x + planeUpNorm.y * myVelocity.y + planeUpNorm.z * myVelocity.z;
	    myVelocity.x = mag;

	    // normalize
	    mag = myVelocity.x * myVelocity.x + myVelocity.y * myVelocity.y;
	    mag = sqrt(mag);
	    if(mag > 0.0f)
	    {
		myVelocity.x = myVelocity.x / mag;
		myVelocity.y = myVelocity.y / mag;
	    }

	    unsigned short data[2];
	    data[0] = __float2half_rn(myVelocity.x);
	    data[1] = __float2half_rn(myVelocity.y);
	    surf2Dwrite(*((uchar4*)data), velSurface, texelIndex.x * 4, texelIndex.y);
	}
    }
}

#define ROUNDING_ADDITION 0.0001f

__global__ void licKernel(int width, int height, float length)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    //unsigned int x = 512;
    //unsigned int y = 512;

    //printf("X: %d Y: %d w: %d h: %d\n",x,y,width,height);

    //if (x < width && y < height) {
	/*unsigned short data[2];
	//data[0] = __float2half_rn(tex2D(noiseTex,x,y));
	data[0] = __float2half_rn(0.0f);
	data[1] = __float2half_rn(1.0f);
	surf2Dwrite(*((uchar4*)data), outSurface, x * 4, y);*/

	/*unsigned short data[2];
	// Read from input surface
	surf2Dread(((uchar4*)data), velSurface, x * 4, y);
	float2 velocity;
	velocity.x = __half2float(data[0]);
	velocity.y = __half2float(data[1]);

	printf("Coord x: %d y: %d Velocity: %f %f\n",x,y,velocity.x,velocity.y);*/
    //}

    float pixelVal = 0.0f;
    float pixelDeb = 0.0f;
    // one thread per pixel
    if (x < width && y < height)
    {
	float2 position;
	float2 weightTotal;

	// init to pixel center
	position.x = x + 0.5f;
	position.y = y + 0.5f;

	weightTotal.x = weightTotal.y = 0.0f;

	int maxIterations = length * 100.0f;
	// forward path
	for(int i = 0; i < maxIterations; ++i)
	{
	    //printf("Position: %f %f\n",position.x,position.y);

	    unsigned short data[2];
	    // Read from input surface
	    surf2Dread(((uchar4*)data), velSurface, ((int)position.x) * 4, ((int)position.y));
	    float2 velocity;
	    velocity.x = __half2float(data[0]);
	    velocity.y = __half2float(data[1]);

	    //printf("Velocity: %f %f\n",velocity.x,velocity.y);

	    // find delta s
	    float top,bottom,left,right;
	    if(velocity.y == 0.0f)
	    {
		//pixelDeb = 1.0f;
		top = bottom = FLT_MAX;
	    }
	    else
	    {
		top = (ceilf(position.y) - position.y) / velocity.y;
		bottom = (floorf(position.y) - position.y) / velocity.y;
	    }

	    if(velocity.x == 0.0f)
	    {
		//pixelDeb = 1.0f;
		left = right = FLT_MAX;
	    }
	    else
	    {
		right = (ceilf(position.x) - position.x) / velocity.x;
		left = (floorf(position.x) - position.x) / velocity.x;
	    }
	    
	    //printf("T: %f B: %f R: %f L: %f\n",top,bottom,right,left);

	    float deltas = FLT_MAX;
	    
	    if(top > 0.0f && top < deltas)
	    {
		deltas = top;
	    }
	    if(bottom > 0.0f && bottom < deltas)
	    {
		deltas = bottom;
	    }
	    if(right > 0.0f && right < deltas)
	    {
		deltas = right;
	    }
	    if(left > 0.0f && left < deltas)
	    {
		deltas = left;
	    }

	    //printf("DeltaS: %f\n",deltas);

	    // zero velocity, going nowhere
	    if(deltas == FLT_MAX)
	    {
		//pixelDeb = 1.0f;
		//printf("FLT_MAX break\n");
		break;
	    }

	    // make sure we hit the next cell
	    deltas = deltas + ROUNDING_ADDITION;

	    // add weight/value

	    // end of line
	    if(weightTotal.x + deltas > length)
	    {
		//pixelDeb = 1.0f;
		/*if(weightTotal.x == 0.0f)
		{
		    printf("Pos ds: %f\n",deltas);
		    printf("T: %f B: %f R: %f L: %f\n",top,bottom,right,left);
		    printf("Velocity: %f %f\n",velocity.x,velocity.y);
		}*/
		//printf("weightTotal break\n");
		break;
	    }

	    weightTotal.x = weightTotal.x + deltas;

	    //printf("WeightTotal: %f\n",weightTotal.x);

	    // add weighted noise
	    pixelVal = pixelVal + (deltas * tex2D(noiseTex,(int)position.x,(int)position.y));
	    //printf("PixelVal: %f\n",pixelVal);

	    // find next point
	    position.x = position.x + deltas * velocity.x;
	    position.y = position.y + deltas * velocity.y;

	    // out of grid
	    if(position.x < 0.0f || position.x >= width || position.y < 0.0f || position.y >= height)
	    {
		//printf("Out of grid break\n");
		break;
	    }
	}

	position.x = x + 0.5f;
	position.y = y + 0.5f;

	// backwards path
	for(int i = 0; i < maxIterations; ++i)
	{
	    //printf("Position: %f %f\n",position.x,position.y);

	    unsigned short data[2];
	    // Read from input surface
	    surf2Dread(((uchar4*)data), velSurface, ((int)position.x) * 4, ((int)position.y));
	    float2 velocity;
	    velocity.x = __half2float(data[0]);
	    velocity.y = __half2float(data[1]);

	    //printf("Velocity: %f %f\n",velocity.x,velocity.y);

	    // find delta s
	    float top,bottom,left,right;
	    if(velocity.y == 0.0f)
	    {
		//pixelDeb = 1.0f;
		top = bottom = FLT_MAX;
	    }
	    else
	    {
		top = (ceilf(position.y) - position.y) / -velocity.y;
		bottom = (floorf(position.y) - position.y) / -velocity.y;
	    }

	    if(velocity.x == 0.0f)
	    {
		//pixelDeb = 1.0f;
		left = right = FLT_MAX;
	    }
	    else
	    {
		right = (ceilf(position.x) - position.x) / -velocity.x;
		left = (floorf(position.x) - position.x) / -velocity.x;
	    }
	    
	    //printf("T: %f B: %f R: %f L: %f\n",top,bottom,right,left);

	    float deltas = FLT_MAX;
	    
	    if(top > 0.0f && top < deltas)
	    {
		deltas = top;
	    }
	    if(bottom > 0.0f && bottom < deltas)
	    {
		deltas = bottom;
	    }
	    if(right > 0.0f && right < deltas)
	    {
		deltas = right;
	    }
	    if(left > 0.0f && left < deltas)
	    {
		deltas = left;
	    }

	    //printf("DeltaS: %f\n",deltas);

	    // zero velocity, going nowhere
	    if(deltas == FLT_MAX)
	    {
		//pixelDeb = 1.0f;
		//printf("FLT_MAX break\n");
		break;
	    }

	    // make sure we hit the next cell
	    deltas = deltas + ROUNDING_ADDITION;

	    // add weight/value

	    // end of line
	    if(weightTotal.y + deltas > length)
	    {
		//pixelDeb = 1.0f;
		/*if(weightTotal.y == 0.0f)
		{
		    printf("Neg ds: %f\n",deltas);
		    printf("T: %f B: %f R: %f L: %f\n",top,bottom,right,left);
		    printf("Velocity: %f %f\n",velocity.x,velocity.y);
		}*/
		//printf("weightTotal break\n");
		break;
	    }

	    weightTotal.y = weightTotal.y + deltas;

	    //printf("WeightTotal: %f\n",weightTotal.x);

	    // add weighted noise
	    pixelVal = pixelVal + (deltas * tex2D(noiseTex,(int)position.x,(int)position.y));
	    //printf("PixelVal: %f\n",pixelVal);

	    // find next point
	    position.x = position.x + deltas * -velocity.x;
	    position.y = position.y + deltas * -velocity.y;

	    // out of grid
	    if(position.x < 0.0f || position.x >= width || position.y < 0.0f || position.y >= height)
	    {
		//printf("Out of grid break\n");
		break;
	    }
	}

	if((weightTotal.x + weightTotal.y) > 0.0f)
	{
	    pixelVal = pixelVal / (weightTotal.x + weightTotal.y);
	    //pixelVal = 0.0f;
	}
	else
	{
	    //pixelVal = 1.0f;
	    //pixelDeb = 1.0f;
	    //printf("Wx: %f Wy: %f\n",weightTotal.x,weightTotal.y);
	}

	//printf("Writing pixelVal: %f\n",pixelVal);

	// set pixel value
	unsigned short data[2];
	data[0] = __float2half_rn(pixelVal);
	data[1] = __float2half_rn(pixelDeb);
        surf2Dwrite(*((uchar4*)data), outSurface, x * 4, y);
    }
}

__global__ void makeTetListKernel(unsigned int * tetList, unsigned int * numTets, int totalTets, uint4 * ind, float3 * verts)
{
    int tetid = blockIdx.x*blockDim.x + threadIdx.x;
    if(tetid >= totalTets)
    {
	return;
    }

    /*if(tetid == 0)
    {
	printf("PlaneUp %f %f %f\n",planeUp.x,planeUp.y,planeUp.z);
	printf("PlaneUpN %f %f %f\n",planeUpNorm.x,planeUpNorm.y,planeUpNorm.z);
	printf("PlaneRight %f %f %f\n",planeRight.x,planeRight.y,planeRight.z);
	printf("PlaneRightN %f %f %f\n",planeRightNorm.x,planeRightNorm.y,planeRightNorm.z);
	printf("PlanePoint %f %f %f\n",planePoint.x,planePoint.y,planePoint.z);
	printf("PlaneNormal %f %f %f\n",planeNormal.x,planeNormal.y,planeNormal.z);
	printf("BasisMat: %f %f %f %f %f %f %f %f %f\n",invBasisMat[0],invBasisMat[1],invBasisMat[2],invBasisMat[3],invBasisMat[4],invBasisMat[5],invBasisMat[6],invBasisMat[7],invBasisMat[8]);
    }*/

    float4 tetpoints[4];
    *((float3*)&tetpoints[0]) = verts[ind[tetid].x];
    *((float3*)&tetpoints[1]) = verts[ind[tetid].y];
    *((float3*)&tetpoints[2]) = verts[ind[tetid].z];
    *((float3*)&tetpoints[3]) = verts[ind[tetid].w];

    //printf("Point0 %f %f %f\n",tetpoints[0].x,tetpoints[0].y,tetpoints[0].z);
    //printf("Point1 %f %f %f\n",tetpoints[1].x,tetpoints[1].y,tetpoints[1].z);
    //printf("Point2 %f %f %f\n",tetpoints[2].x,tetpoints[2].y,tetpoints[2].z);
    //printf("Point3 %f %f %f\n",tetpoints[3].x,tetpoints[3].y,tetpoints[3].z);

    // find viewing plane distance
    tetpoints[0].w = (tetpoints[0].x - planePoint.x) * planeNormal.x + (tetpoints[0].y - planePoint.y) * planeNormal.y + (tetpoints[0].z - planePoint.z) * planeNormal.z;
    tetpoints[1].w = (tetpoints[1].x - planePoint.x) * planeNormal.x + (tetpoints[1].y - planePoint.y) * planeNormal.y + (tetpoints[1].z - planePoint.z) * planeNormal.z;
    tetpoints[2].w = (tetpoints[2].x - planePoint.x) * planeNormal.x + (tetpoints[2].y - planePoint.y) * planeNormal.y + (tetpoints[2].z - planePoint.z) * planeNormal.z;
    tetpoints[3].w = (tetpoints[3].x - planePoint.x) * planeNormal.x + (tetpoints[3].y - planePoint.y) * planeNormal.y + (tetpoints[3].z - planePoint.z) * planeNormal.z;

    //printf("Distances: %f %f %f %f\n",tetpoints[0].w,tetpoints[1].w,tetpoints[2].w,tetpoints[3].w);

    // determine if plane passes through tet
    int count = 0;
    if(tetpoints[0].w > 0.0f)
    {
	count++;
    }
    if(tetpoints[1].w > 0.0f)
    {
	count++;
    }
    if(tetpoints[2].w > 0.0f)
    {
	count++;
    }
    if(tetpoints[3].w > 0.0f)
    {
	count++;
    }

    if(count == 0 || count == 4)
    {
	return;
    }

    // project points onto plane and find basis values
    float3 projpoint;
    float2 basisMin;
    float2 basisMax;
    float tempx;
    projpoint.x = tetpoints[0].x - (planeNormal.x * tetpoints[0].w) - planePoint.x;
    projpoint.y = tetpoints[0].y - (planeNormal.y * tetpoints[0].w) - planePoint.y;
    projpoint.z = tetpoints[0].z - (planeNormal.z * tetpoints[0].w) - planePoint.z;

    //printf("projpoint: %f %f %f\n",projpoint.x,projpoint.y,projpoint.z);

    //tempx = projpoint.x * invBasisMat[0] + projpoint.y * invBasisMat[3] + projpoint.z * invBasisMat[6];
    //projpoint.y = projpoint.x * invBasisMat[1] + projpoint.y * invBasisMat[4] + projpoint.z * invBasisMat[7];
    tempx = (projpoint.x * planeUpNorm.x + projpoint.y * planeUpNorm.y + projpoint.z * planeUpNorm.z) / basisLength;
    projpoint.y = (projpoint.x * planeRightNorm.x + projpoint.y * planeRightNorm.y + projpoint.z * planeRightNorm.z) / basisLength;

    //printf("basis %f %f\n",tempx,projpoint.y);

    basisMin.x = basisMax.x = tempx;
    basisMin.y = basisMax.y = projpoint.y;
    
    projpoint.x = tetpoints[1].x - (planeNormal.x * tetpoints[1].w) - planePoint.x;
    projpoint.y = tetpoints[1].y - (planeNormal.y * tetpoints[1].w) - planePoint.y;
    projpoint.z = tetpoints[1].z - (planeNormal.z * tetpoints[1].w) - planePoint.z;

    //tempx = projpoint.x * invBasisMat[0] + projpoint.y * invBasisMat[3] + projpoint.z * invBasisMat[6];
    //projpoint.y = projpoint.x * invBasisMat[1] + projpoint.y * invBasisMat[4] + projpoint.z * invBasisMat[7];
    tempx = (projpoint.x * planeUpNorm.x + projpoint.y * planeUpNorm.y + projpoint.z * planeUpNorm.z) / basisLength;
    projpoint.y = (projpoint.x * planeRightNorm.x + projpoint.y * planeRightNorm.y + projpoint.z * planeRightNorm.z) / basisLength;


    //printf("basis %f %f\n",tempx,projpoint.y);

    basisMin.x = fminf(basisMin.x,tempx);
    basisMax.x = fmaxf(basisMax.x,tempx);
    basisMin.y = fminf(basisMin.y,projpoint.y);
    basisMax.y = fmaxf(basisMax.y,projpoint.y);

    projpoint.x = tetpoints[2].x - (planeNormal.x * tetpoints[2].w) - planePoint.x;
    projpoint.y = tetpoints[2].y - (planeNormal.y * tetpoints[2].w) - planePoint.y;
    projpoint.z = tetpoints[2].z - (planeNormal.z * tetpoints[2].w) - planePoint.z;

    //tempx = projpoint.x * invBasisMat[0] + projpoint.y * invBasisMat[3] + projpoint.z * invBasisMat[6];
    //projpoint.y = projpoint.x * invBasisMat[1] + projpoint.y * invBasisMat[4] + projpoint.z * invBasisMat[7];
    tempx = (projpoint.x * planeUpNorm.x + projpoint.y * planeUpNorm.y + projpoint.z * planeUpNorm.z) / basisLength;
    projpoint.y = (projpoint.x * planeRightNorm.x + projpoint.y * planeRightNorm.y + projpoint.z * planeRightNorm.z) / basisLength;

    //printf("basis %f %f\n",tempx,projpoint.y);

    basisMin.x = fminf(basisMin.x,tempx);
    basisMax.x = fmaxf(basisMax.x,tempx);
    basisMin.y = fminf(basisMin.y,projpoint.y);
    basisMax.y = fmaxf(basisMax.y,projpoint.y);

    projpoint.x = tetpoints[3].x - (planeNormal.x * tetpoints[3].w) - planePoint.x;
    projpoint.y = tetpoints[3].y - (planeNormal.y * tetpoints[3].w) - planePoint.y;
    projpoint.z = tetpoints[3].z - (planeNormal.z * tetpoints[3].w) - planePoint.z;

    //tempx = projpoint.x * invBasisMat[0] + projpoint.y * invBasisMat[3] + projpoint.z * invBasisMat[6];
    //projpoint.y = projpoint.x * invBasisMat[1] + projpoint.y * invBasisMat[4] + projpoint.z * invBasisMat[7];
    tempx = (projpoint.x * planeUpNorm.x + projpoint.y * planeUpNorm.y + projpoint.z * planeUpNorm.z) / basisLength;
    projpoint.y = (projpoint.x * planeRightNorm.x + projpoint.y * planeRightNorm.y + projpoint.z * planeRightNorm.z) / basisLength;

    //printf("basis %f %f\n",tempx,projpoint.y);

    basisMin.x = fminf(basisMin.x,tempx);
    basisMax.x = fmaxf(basisMax.x,tempx);
    basisMin.y = fminf(basisMin.y,projpoint.y);
    basisMax.y = fmaxf(basisMax.y,projpoint.y);

    //printf("X: %f %f\n",basisMin.x,basisMax.x);
    //printf("Y: %f %f\n",basisMin.y,basisMax.y);

    /*basisMin.x = ceilf(basisMin.x - 0.5f) + 0.5f;
    basisMin.y = ceilf(basisMin.y - 0.5f) + 0.5f;
    basisMax.x = floorf(basisMax.x - 0.5f) + 0.500001f;
    basisMax.y = floorf(basisMax.y - 0.5f) + 0.500001f;*/

    basisMin.x = floorf(basisMin.x - 0.5f) - 0.5f;
    basisMin.y = floorf(basisMin.y - 0.5f) - 0.5f;
    basisMax.x = ceilf(basisMax.x + 0.5f) + 0.500001f;
    basisMax.y = ceilf(basisMax.y + 0.5f) + 0.500001f;

    //printf("X: %f %f\n",basisMin.x,basisMax.x);
    //printf("Y: %f %f\n",basisMin.y,basisMax.y);

    if(basisMin.x > basisMax.x || basisMin.y > basisMax.y)
    {
	//printf("Basis range exit x: %f %f y: %f %f\n",basisMin.x,basisMax.x,basisMin.y,basisMax.y);
	// no critical points in tet
	return;
    }

    //find if tet is outside the basis bounds
    if(basisMin.x > texXMax || basisMax.x < texXMin || basisMin.y > texYMax || basisMax.y < texYMin)
    {
	//printf("X: %f %f\n",basisMin.x,basisMax.x);
	//printf("Y: %f %f\n",basisMin.y,basisMax.y);
	return;
    }

    unsigned int index = atomicAdd(numTets,(unsigned int)1);
    tetList[index] = tetid;
}
